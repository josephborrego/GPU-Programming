
#include <hip/hip_runtime.h>
__global__
void deviceKernel(int *a, int N)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for (int i = idx; i < N; i += stride)
  {
    a[i] = 1;
  }
}

void hostFunction(int *a, int N)
{
  for (int i = 0; i < N; ++i)
  {
    a[i] = 1;
  }
}

int main()
{

  int N = 2<<24;
  size_t size = N * sizeof(int);
  int *a;
  hipMallocManaged(&a, size);

  /*
   * Conduct experiments to learn more about the behavior of
   * `cudaMallocManaged`.
   *
   * What happens when unified memory is accessed only by the GPU?
   * What happens when unified memory is accessed only by the CPU?
   * What happens when unified memory is accessed first by the GPU then the CPU?
   * What happens when unified memory is accessed first by the CPU then the GPU?
   *
   * Hypothesize about UM behavior, page faulting specificially, before each
   * experiement, and then verify by running `nvprof`.
   */

//What happens when unified memory is accessed only by the CPU?
//160.77ms, 384 page faults
//hostFunction(a, N);

//What happens when unified memory is accessed only by the GPU?
//23.12ms, no page faults
//deviceKernel<<<256, 256>>>(a, N);
//cudaDeviceSynchronize();

//What happens when unified memory is accessed first by the CPU then the GPU?
//161.58ms, 384 page faults
//hostFunction(a, N);
//cudaDeviceSynchronize();

//What happens when unified memory is accessed first by the GPU then the CPU?
//11.37ms, 384 page faults
deviceKernel<<<256, 256>>>(a, N);
hipDeviceSynchronize();
hostFunction(a, N);

  hipFree(a);
}
